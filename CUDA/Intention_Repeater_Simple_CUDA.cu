#include "hip/hip_runtime.h"
/*
Intention Repeater Simple CUDA
by Anthro Teacher, WebGPT and Claude 3 Opus
To compile: nvcc Intention_Repeater_Simple_CUDA.cu -o Intention_Repeater_Simple_CUDA.exe -L/Users/tswee/miniconda3/Library/lib -lz
To run: Intention_Repeater_Simple_CUDA.exe --intent "I am Love." --imem 1 --hashing y --compress n --dur 00:00:10
*/

#include "picosha2.h"
#include "zlib.h"
#include <iostream>
#include <iomanip>
#include <string>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include <csignal>
#include <atomic>

using namespace std;
using namespace std::chrono;

const int ONE_MINUTE = 60;
const int ONE_HOUR = 3600;

string VERSION = "v1.3";

std::atomic<bool> interrupted(false);

void signalHandler(int signum)
{
    //cout << "\nInterrupt signal (" << signum << ") received.\n";
    interrupted.store(true);
}

// CUDA kernel to perform intention repeating and frequency updating
__global__ void intentionRepeaterKernel(const char *intentionMultiplied, unsigned long long int *freq, size_t intentionSize)
{
    unsigned long long int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < intentionSize)
    {
        atomicAdd(freq, 1);
    }
}

std::string compressMessage(const std::string &message)
{
    z_stream zs;
    memset(&zs, 0, sizeof(zs));

    if (deflateInit(&zs, Z_DEFAULT_COMPRESSION) != Z_OK)
    {
        return ""; // Compression initialization failed
    }

    zs.next_in = (Bytef *)message.data();
    zs.avail_in = message.size();

    std::string compressed;
    char outbuffer[32768]; // Output buffer
    int ret;
    do
    {
        zs.next_out = reinterpret_cast<Bytef *>(outbuffer);
        zs.avail_out = sizeof(outbuffer);

        ret = deflate(&zs, Z_FINISH);

        if (compressed.size() < zs.total_out)
        {
            compressed.append(outbuffer, zs.total_out - compressed.size());
        }
    } while (ret == Z_OK);

    deflateEnd(&zs);

    if (ret != Z_STREAM_END)
    {
        return ""; // Compression failed
    }

    return compressed;
}

string FormatTime(long long seconds)
{
    int hours = seconds / ONE_HOUR;
    int minutes = (seconds % ONE_HOUR) / ONE_MINUTE;
    int secs = seconds % ONE_MINUTE;

    ostringstream oss;
    oss << setw(2) << setfill('0') << hours << ":"
        << setw(2) << setfill('0') << minutes << ":"
        << setw(2) << setfill('0') << secs;

    return oss.str();
}

void print_help()
{
    cout << "Intention Repeater Simple CUDA by Anthro Teacher." << endl;
    cout << "Repeats your intention millions of times per second " << endl;
    cout << "in computer memory, to aid in manifestation." << endl;
    cout << "Optional Flags:" << endl;
    cout << " a) --intent or -i, example: --intent \"I am Love.\" [The Intention]" << endl;
    cout << " b) --imem or -m, example: --imem 2 [GB of RAM to Use]" << endl;
    cout << "    --imem 0 to disable Intention Multiplying" << endl;
    cout << " c) --dur or -d, example: --dur 00:01:00 [Running Duration HH:MM:SS]" << endl;
    cout << " d) --hashing or -h, example: --hashing y [Use Hashing]" << endl;
    cout << " e) --compress or -c, example: --compress y [Use Compression]" << endl;
    cout << " f) --file or -f, example: --file \"intentions.txt\" [File to Read Intentions From]" << endl;
    cout << " g) --help or -? [This help]" << endl;
}

string DisplaySuffix(const string &num, int power, const string &designator)
{
    const string suffixArray = designator == "Iterations" ? " kMBTqQsSOND" : " kMGTPEZYR";
    size_t index = power / 3;
    char suffix = index < suffixArray.length() ? suffixArray[index] : ' ';
    string result = num.substr(0, power % 3 + 1) + "." + num.substr(power % 3 + 1, 3) + suffix;
    return result;
}

string FindSum(const string &a, const string &b)
{
    string result;
    int carry = 0;

    int i = a.size() - 1;
    int j = b.size() - 1;

    while (i >= 0 || j >= 0 || carry > 0)
    {
        int sum = carry;

        if (i >= 0)
        {
            sum += a[i] - '0';
            --i;
        }

        if (j >= 0)
        {
            sum += b[j] - '0';
            --j;
        }

        result.push_back(sum % 10 + '0');
        carry = sum / 10;
    }

    reverse(result.begin(), result.end());
    return result;
}

string MultiplyStrings(const string &num1, const string &num2)
{
    int len1 = num1.size();
    int len2 = num2.size();
    vector<int> result(len1 + len2, 0);

    for (int i = len1 - 1; i >= 0; --i)
    {
        for (int j = len2 - 1; j >= 0; --j)
        {
            int mul = (num1[i] - '0') * (num2[j] - '0');
            int sum = mul + result[i + j + 1];

            result[i + j + 1] = sum % 10;
            result[i + j] += sum / 10;
        }
    }

    string resultStr;
    for (int num : result)
    {
        if (!(resultStr.empty() && num == 0))
        {
            resultStr.push_back(num + '0');
        }
    }

    return resultStr.empty() ? "0" : resultStr;
}

void readFileContents(const std::string &filename,
                      std::string &intention_file_contents)
{
    std::ifstream file(filename, std::ios::binary);
    if (!file)
    {
        std::cerr << "File not found" << std::endl;
        std::exit(EXIT_FAILURE); // Terminate the program
    }

    std::ostringstream buffer;
    char ch;
    while (file.get(ch))
    {
        if (ch != '\0')
        {
            buffer.put(ch);
        }
    }

    intention_file_contents = buffer.str();
    file.close();
}

int main(int argc, char **argv)
{
    std::cout << "Intention Repeater Simple CUDA " << VERSION << endl;
    std::cout << "by Anthro Teacher and WebGPT" << endl
              << endl;

    std::signal(SIGINT, signalHandler);
    string intention = "", param_intent = "X", param_imem = "X", param_duration = "INFINITY", param_hashing = "X";
    string useHashing, useCompression, param_compress = "X", param_file = "X", intention_display = "", file_contents="", intention_value="";
    int numGBToUse = 1;

    for (int i = 1; i < argc; i++)
    {
        if (!strcmp(argv[i], "-?") || !strcmp(argv[i], "--help") || !strcmp(argv[i], "/?"))
        {
            print_help();
            exit(EXIT_SUCCESS);
        }
        else if (!strcmp(argv[i], "-i") || !strcmp(argv[i], "--intent"))
        {
            param_intent = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-m") || !strcmp(argv[i], "--imem"))
        {
            param_imem = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-d") || !strcmp(argv[i], "--dur"))
        {
            param_duration = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-h") || !strcmp(argv[i], "--hashing"))
        {
            param_hashing = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-c") || !strcmp(argv[i], "--compress"))
        {
            param_compress = argv[i + 1];
        }
        else if (!strcmp(argv[i], "-f") || !strcmp(argv[i], "--file"))
        {
            param_file = argv[i + 1];
        }
    }

    if (param_file != "X")
    {
        // Open param_intent file and read the full file contents into intention
        readFileContents(param_file, file_contents);
        intention_display = "Contents of: " + param_file;
    }
    
    if (param_intent == "X")
    {
        while (!interrupted)
        {
            std::cout << "Enter your Intention: ";
            if (!std::getline(std::cin, intention))
            {
                // If getline fails (e.g., due to an interrupt), break out of the loop immediately
                interrupted.store(true); // Ensure the flag is set if not already
                return 0;
            }

            if (!intention.empty())
            {
                break; // Successfully got an intention, exit the loop
            }
            else if (!interrupted)
            {
                // Only show the message if we're not interrupted
                std::cout << "The intention cannot be empty. Please try again.\n";
            }
        }
        intention_display = intention;
        intention_value = intention;
    }
    else
    {
        intention = param_intent;
        intention_value = intention;
        intention_display = param_intent;
    }

    if (param_file != "X")
    {
        // Keep adding intention_value onto intention until its length is >= length of file_contents
        while (intention.length() < file_contents.length())
        {
            intention += intention_value;
        }
        intention += file_contents;
        intention_display += " (" + param_file + ")";
    }

    if (!interrupted)
    {
        if (param_imem == "X")
        {
            std::cout << "GB RAM to Use [Default 1]: ";
            string input;
            if (!std::getline(std::cin, input))
            {
                // If getline fails due to interruption
                interrupted.store(true); // Ensure the flag is properly set
                if (interrupted)
                {
                    // std::cerr << "Interrupted. Exiting configuration.\n";
                    return 0; // Exit or handle as necessary
                }
            }

            if (!input.empty())
            {
                try
                {
                    numGBToUse = stoi(input);
                }
                catch (const std::invalid_argument &e)
                {
                    // std::cerr << "Invalid input, using default of 1 GB.\n";
                    numGBToUse = 1;
                }
                catch (const std::out_of_range &e)
                {
                    // std::cerr << "Input out of range, using default of 1 GB.\n";
                    numGBToUse = 1;
                }
            }
        }
        else
        {
            numGBToUse = stoi(param_imem);
        }
    }

    if (!interrupted && param_hashing == "X")
    {
        std::cout << "Use Hashing (y/N): ";
        if (!std::getline(std::cin, useHashing))
        {
            interrupted.store(true);
            if (interrupted)
            {
                // std::cerr << "Interrupted during hashing input. Exiting configuration.\n";
                return 0;
            }
        }
        transform(useHashing.begin(), useHashing.end(), useHashing.begin(), ::tolower);
    }
    else if (!interrupted)
    {
        useHashing = param_hashing;
        // Convert useHashing to lowercase
        transform(useHashing.begin(), useHashing.end(), useHashing.begin(), ::tolower);
    }

    if (!interrupted && param_compress == "X")
    {
        std::cout << "Use Compression (y/N): ";
        if (!std::getline(std::cin, useCompression))
        {
            interrupted.store(true);
            if (interrupted)
            {
                // std::cerr << "Interrupted during compression input. Exiting configuration.\n";
                return 0;
            }
        }
        transform(useCompression.begin(), useCompression.end(), useCompression.begin(), ::tolower);
    }
    else if (!interrupted)
    {
        useCompression = param_compress;
        // Convert useCompression to lowercase
        transform(useCompression.begin(), useCompression.end(), useCompression.begin(), ::tolower);
    }

    string intentionMultiplied, intentionHashed;
    size_t ramSize = 1024ULL * 1024 * 512 * numGBToUse;
    size_t multiplier = 0, hashMultiplier = 1;

    if (!interrupted)
    {
        std::cout << "Loading..." << string(10, ' ') << "\r" << flush;
    }
    else
    {
        return 0;
    }

    if (numGBToUse > 0)
    {
        while (intentionMultiplied.length() < ramSize)
        {
            intentionMultiplied += intention;
            ++multiplier;
        }
    }
    else
    {
        intentionMultiplied = intention;
        multiplier = 1;
    }

    if (useHashing == "y" || useHashing == "yes")
    {
        intentionHashed = picosha2::hash256_hex_string(intentionMultiplied);
        intentionMultiplied.clear();
        if (numGBToUse > 0)
        {
            while (intentionMultiplied.length() < ramSize)
            {
                intentionMultiplied += intentionHashed;
                ++hashMultiplier;
            }
        }
        else
        {
            intentionMultiplied = intentionHashed;
            hashMultiplier = 1;
        }
    }
    else
    {
        hashMultiplier = 1;
    }

    if (useCompression == "y" || useCompression == "yes")
    {
        intentionMultiplied = compressMessage(intentionMultiplied);
    }

    // Allocate memory on the GPU for intentionMultiplied and freq
    char *d_intentionMultiplied;
    unsigned long long int *d_freq;
    hipMalloc(&d_intentionMultiplied, intentionMultiplied.size());
    hipMalloc(&d_freq, sizeof(unsigned long long int));

    // Copy intentionMultiplied to the GPU
    hipMemcpy(d_intentionMultiplied, intentionMultiplied.c_str(), intentionMultiplied.size(), hipMemcpyHostToDevice);

    string totalIterations = "0", totalFreq = "0";
    unsigned long long freq = 0, seconds = 0;

    while (!interrupted)
    {
        auto start = high_resolution_clock::now();
        auto end = start + chrono::duration_cast<chrono::seconds>(chrono::seconds(1));

        // Set freq to 0 on the GPU
        hipMemset(d_freq, 0, sizeof(unsigned long long int));

        while (high_resolution_clock::now() < end)
        {
            // Launch the CUDA kernel for intention repeating and frequency updating
            int blockSize = 256;
            int numBlocks = (intentionMultiplied.size() + blockSize - 1) / blockSize;
            intentionRepeaterKernel<<<numBlocks, blockSize>>>(d_intentionMultiplied, d_freq, intentionMultiplied.size());

            // Wait for the GPU to finish before accessing on host
            hipDeviceSynchronize();

            // Copy the updated freq back to the CPU
            hipMemcpy(&freq, d_freq, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
        }

        totalFreq = MultiplyStrings(to_string(freq), to_string(multiplier));
        totalFreq = MultiplyStrings(totalFreq, to_string(hashMultiplier));
        totalIterations = FindSum(totalIterations, totalFreq);

        int digits = totalIterations.length();
        int freqDigits = totalFreq.length();
        ++seconds;
        freq = 0;

        std::cout << "[" + FormatTime(seconds) + "] "
                  << " (" << DisplaySuffix(totalIterations, digits - 1, "Iterations")
                  << " / " << DisplaySuffix(totalFreq, freqDigits - 1, "Frequency") << "Hz): " << intention_display
                  << string(5, ' ') << "\r" << flush;
        if (param_duration == FormatTime(seconds))
        {
            interrupted = true;
        }

        if (interrupted)
        {
            break;
        }
    }

    std::cout << endl;
    // Free allocated memory on the GPU
    hipFree(d_intentionMultiplied);
    hipFree(d_freq);

    return 0;
}